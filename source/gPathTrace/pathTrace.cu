#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017, NVIDIA Corporation. 
//
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer 
//    in the documentation and/or  other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may be used to endorse or promote products derived 
//    from this software without specific prior written permission.
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING,
// BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL 
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE 
// OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Version 1.0: Sergen Eren, 26/3/2019
//----------------------------------------------------------------------------------
// 
// File: Custom path trace kernel: 
//       Contents are modified from "cuda_gvdb_raycast.cuh" in gvdb library
//
//-----------------------------------------------




#include <stdio.h>
#include "cuda_math.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


typedef unsigned char		uchar;
typedef unsigned int		uint;
typedef unsigned short		ushort;
typedef unsigned long		ulong;
typedef unsigned long long	uint64;

//-------------------------------- GVDB Data Structure
#define CUDA_PATHWAY
#include "cuda_gvdb_scene.cuh"		// GVDB Scene
#include "cuda_gvdb_nodes.cuh"		// GVDB Node structure
#include "cuda_gvdb_geom.cuh"		// GVDB Geom helpers
#include "cuda_gvdb_dda.cuh"		// GVDB DDA 



// gvdbBrickFunc ( gvdb, channel, nodeid, t, pos, dir, pstep, hit, norm, clr )
typedef void(*gvdbBrickFunc_t)(VDBInfo*, uchar, int, float3, float3, float3, float3&, float3&, float3&, float4&);

#define MAXLEV			5
#define MAX_ITER		256
#define EPS				0.0001

#define LO		0
#define	MID		1.0
#define	HI		2.0
#define M_PI       3.14159265358979323846   // pi


// Helper functions 
inline __device__ float getLinearDepth(float* depthBufFloat)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;					// Pixel coordinates
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float z = depthBufFloat[(SCN_HEIGHT - 1 - y) * SCN_WIDTH + x];	// Get depth value
	float n = scn.camnear;
	float f = scn.camfar;
	return (-n * f / (f - n)) / (z - (f / (f - n)));				// Return linear depth
}

inline __device__ uchar4 getColor(VDBInfo* gvdb, uchar chan, float3 p)
{
	return tex3D<uchar4>(gvdb->volIn[chan], (int)p.x, (int)p.y, (int)p.z);
}
inline __device__ float4 getColorF(VDBInfo* gvdb, uchar chan, float3 p)
{
	return make_float4(tex3D<uchar4>(gvdb->volIn[chan], (int)p.x, (int)p.y, (int)p.z));
}



// Brick sampler

__device__ void customRayDeepBrick(VDBInfo* gvdb, uchar chan, int nodeid, float3 t, float3 pos, float3 dir, float3& pstep, float3& hit, float3& norm, float4& clr)
{
	float3 vmin;
	VDBNode* node = getNode(gvdb, 0, nodeid, &vmin);			// Get the VDB leaf node		

	//t.x = SCN_PSTEP * ceil( t.x / SCN_PSTEP );						// start on sampling wavefront	

	float3 o = make_float3(node->mValue);					// atlas sub-volume to trace
	float3 wp = pos + t.x*dir;
	float3 p = (wp - vmin) / gvdb->vdel[0];					// sample point in index coords	
	float3 wpt = SCN_PSTEP * dir * gvdb->vdel[0];					// world increment
	float4 val = make_float4(0, 0, 0, 0);
	float4 hclr;
	int iter = 0;
	float dt = length(SCN_PSTEP*dir*gvdb->vdel[0]);

	// record front hit point at first significant voxel
	if (hit.x == 0) hit.x = t.x; // length(wp - pos);

	// skip empty voxels
	for (iter = 0; val.w < SCN_MINVAL && iter < MAX_ITER && p.x >= 0 && p.y >= 0 && p.z >= 0 && p.x < gvdb->res[0] && p.y < gvdb->res[0] && p.z < gvdb->res[0]; iter++) {
		val.w = transfer(gvdb, tex3D<float>(gvdb->volIn[chan], p.x + o.x, p.y + o.y, p.z + o.z)).w;
		p += SCN_PSTEP * dir;
		wp += wpt;
		t.x += dt;
	}

	for (; iter < MAX_ITER && p.x >= 0 && p.y >= 0 && p.z >= 0 && p.x < gvdb->res[0] && p.y < gvdb->res[0] && p.z < gvdb->res[0]; iter++) {

		if (clr.x > 1 || clr.y > 1 || clr.z > 1 || clr.w > 1) return;
		val = transfer(gvdb, tex3D<float>(gvdb->volIn[chan], p.x + o.x, p.y + o.y, p.z + o.z));
		clr += val; 
		p += SCN_PSTEP * dir;
		wp += wpt;
		t.x += dt;

	}


	hit.y = t.x;  // length(wp - pos);

}

__device__ void myRayCast(VDBInfo* gvdb, uchar chan, float3 pos, float3 dir, float3& hit, float3& norm, float4& clr, gvdbBrickFunc_t brickFunc)
{
	int		nodeid[MAXLEV];					// level variables
	float	tMax[MAXLEV];
	int		b;

	// GVDB - Iterative Hierarchical 3DDA on GPU
	float3 vmin;
	int lev = gvdb->top_lev;
	nodeid[lev] = 0;		// rootid ndx
	float3 t = rayBoxIntersect(pos, dir, gvdb->bmin, gvdb->bmax);	// intersect ray with bounding box	
	VDBNode* node = getNode(gvdb, lev, nodeid[lev], &vmin);			// get root VDB node	
	if (t.z == NOHIT) return; //TODO:implement texture lookup here

	// 3DDA variables		
	t.x += gvdb->epsilon;
	tMax[lev] = t.y - gvdb->epsilon;
	float3 pStep = make_float3(isign3(dir));
	float3 p, tDel, tSide, mask;
	int iter;

	PREPARE_DDA

		for (iter = 0; iter < MAX_ITER && lev > 0 && lev <= gvdb->top_lev && p.x >= 0 && p.y >= 0 && p.z >= 0 && p.x <= gvdb->res[lev] && p.y <= gvdb->res[lev] && p.z <= gvdb->res[lev]; iter++) {

			NEXT_DDA

				// depth buffer test [optional]
				if (SCN_DBUF != 0x0) {
					if (t.x > getLinearDepth(SCN_DBUF)) {
						hit.z = 0;
						return;
					}
				}

			// node active test
			b = (((int(p.z) << gvdb->dim[lev]) + int(p.y)) << gvdb->dim[lev]) + int(p.x);	// bitmaskpos
			if (isBitOn(gvdb, node, b)) {							// check vdb bitmask for voxel occupancy						
				if (lev == 1) {									// enter brick function..
					nodeid[0] = getChild(gvdb, node, b);
					t.x += gvdb->epsilon;
					(*brickFunc) (gvdb, chan, nodeid[0], t, pos, dir, pStep, hit, norm, clr);
					if (clr.w <= 0) {
						clr.w = 0;
						return;
					}			// deep termination				
					if (hit.z != NOHIT) return;						// surface termination												

					STEP_DDA										// leaf node empty, step DDA
					//t.x = hit.y;				
					//PREPARE_DDA

				}
				else {
					lev--;											// step down tree
					nodeid[lev] = getChild(gvdb, node, b);				// get child 
					node = getNode(gvdb, lev, nodeid[lev], &vmin);	// child node
					t.x += gvdb->epsilon;										// make sure we start inside child
					tMax[lev] = t.y - gvdb->epsilon;							// t.x = entry point, t.y = exit point							
					PREPARE_DDA										// start dda at next level down
				}
			}
			else {
				STEP_DDA											// empty voxel, step DDA
			}
			while (t.x > tMax[lev] && lev <= gvdb->top_lev) {
				lev++;												// step up tree
				if (lev <= gvdb->top_lev) {
					node = getNode(gvdb, lev, nodeid[lev], &vmin);
					PREPARE_DDA										// restore dda at next level up
				}
			}
		}
}


extern "C" __global__ void pathTrace(VDBInfo* gvdb, uchar chan, uchar4* outBuf) {


	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= scn.width || y >= scn.height) return;
	
	float3 rdir = normalize(getViewRay((float(x) + 0.5) / scn.width, (float(y) + 0.5) / scn.height));
	float3 hit = make_float3(NOHIT, NOHIT, NOHIT);
	float4 clr = make_float4(0.1f, 0.1f, 0.1f, 0.1f);
	float3 norm;
	
	float4 density = make_float4(0,0,0,0); 

	myRayCast(gvdb, chan, scn.campos, rdir, hit, norm, clr, customRayDeepBrick);
	
	
	outBuf[y*scn.width + x] = make_uchar4(clr.x*255 , clr.y*255, clr.z*255, 1);


}