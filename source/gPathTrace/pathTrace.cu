#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017, NVIDIA Corporation. 
//
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer 
//    in the documentation and/or  other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may be used to endorse or promote products derived 
//    from this software without specific prior written permission.
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING,
// BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL 
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE 
// OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Version 1.0: Sergen Eren, 26/3/2019
//----------------------------------------------------------------------------------
// 
// File: Custom path trace kernel: 
//       Performs a custom ray marching inside volume
//
//-----------------------------------------------

#include <stdio.h>
#include "cuda_math.cuh"
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h>


typedef unsigned char		uchar;
typedef unsigned int		uint;
typedef unsigned short		ushort;
typedef unsigned long		ulong;
typedef unsigned long long	uint64;

//-------------------------------- GVDB Data Structure
#define CUDA_PATHWAY
#include "cuda_gvdb_scene.cuh"		// GVDB Scene
#include "cuda_gvdb_nodes.cuh"		// GVDB Node structure
#include "cuda_gvdb_geom.cuh"		// GVDB Geom helpers
#include "cuda_gvdb_dda.cuh"		// GVDB DDA 

#define MAXLEV			5
#define MAX_ITER		256
#define EPS				0.0001

#define LO				0
#define	MID				1.0
#define	HI				2.0
#define M_PI			3.14159265358979323846f   // pie
#define INV_4PI			1 / 4 * M_PI

// Helper functions 

inline __device__ uchar4 getColor(VDBInfo* gvdb, uchar chan, float3 p)
{
	return tex3D<uchar4>(gvdb->volIn[chan], (int)p.x, (int)p.y, (int)p.z);
}
inline __device__ float4 getColorF(VDBInfo* gvdb, uchar chan, float3 p)
{
	return make_float4(tex3D<uchar4>(gvdb->volIn[chan], (int)p.x, (int)p.y, (int)p.z));
}
inline __device__ float3 exp3(float3 val)
{
	float3 tmp = make_float3(exp(val.x), exp(val.y), exp(val.z));
	return tmp;
}

#define EPSTEST(a,b,c)	(a>b-c && a<b+c)
#define VOXEL_EPS	0.0001

//Phase functions 

__device__ float isotropic() {

	return INV_4PI;

}

__device__ float henyey_greenstein(float cos_theta, float g) {

	float denominator = 1 + g * g - 2 * g * cos_theta;

	return INV_4PI * (1 - g * g) / (denominator * sqrtf(denominator));

}

__device__ float double_henyey_greenstein(float cos_theta, float f, float g1, float g2) {

	return (1 - f)*henyey_greenstein(cos_theta, g1) + f * henyey_greenstein(cos_theta, g2);

}

__device__ float schlick(float cos_theta, float k) { // simpler hg phase function Note: -1<k<1   

	float denominator = 1 + k * cos_theta;

	return INV_4PI * (1 - k * k) / (denominator*denominator);

}

__device__ float rayleigh(float cos_sq_theta, float lambda) // rayleigh scattering
{

	return 3 * (1 + cos_sq_theta) / 4 * lambda*lambda*lambda*lambda; // 

}

__device__ float cornette_shanks(float cos_theta, float cos_sq_theta, float g) {

	float first_part = (1 - g * g) / (2 + g * g);
	float second_part = (1 + cos_sq_theta) / pow((1 + g * g - cos_theta), 1.5f);

	return INV_4PI * 1.5f * first_part * second_part;

}
// End phase functions


// Shadow ray marcher
__device__ float3 getShadowTransmittance(float3 pos, float sampledDistance, float stepSizeShadow, float3 extinction) {

	float3 shadow = make_float3(1.0f);
	float3 Ldir = normalize(scn.light_pos - pos);

	for (float tshadow = 0.0f; tshadow < sampledDistance; tshadow += stepSizeShadow) {

		float3 shadowPos = pos + Ldir * tshadow;
		float densityShadow = 1.0f;
		shadow *= exp3(-densityShadow * extinction*stepSizeShadow);
	}

	return shadow;

}

__device__ bool in_brick(VDBInfo* gvdb,  float3 pos) {

	return pos.x >= 0 && pos.y >= 0 && pos.z >= 0 && pos.x < gvdb->res[0] && pos.y < gvdb->res[0] && pos.z < gvdb->res[0];
}
__device__ void RayCast(VDBInfo* gvdb, uchar chan, float3 pos, float3 dir, float3& hit, float4& clr) {

	float3 absorption = 10.0f * make_float3(0.75, 0.5, 0.0);
	float3 scattering = 25.0f * make_float3(0.25, 0.5, 1.0);
	float3 extinction = absorption + scattering;

	float density = 0.0f;

	float3 scatteredLuminance = make_float3(0.0, 0.0, 0.0);
	float3 transmittance = make_float3(1.0);
	float3 L = make_float3(50, 50, 50); // Light color
	float3 color = make_float3(1.0, 0.0, 0.0);
	float stepSize = 0.005f;


	float3 t = rayBoxIntersect(pos, dir, gvdb->bmin, gvdb->bmax);
	if (t.z == NOHIT) return;

	float3 wpos = pos + dir * t.x; //get world position at first intersection 
	wpos += dir * 0.001; // add epsilon

	for (float f = t.x; f < t.y; f += stepSize) {

		if (transmittance.x < 0.1f) break; // no need to trace further

		//brick node variables 
		float3 vmin; //root pos of brick node
		uint64 nodeid; // brick id 
		float3 offset; // brick offset
		float3 vdel; // i.e. voxel size 

		VDBNode* brick_node = getNodeAtPoint(gvdb, wpos + dir * stepSize , &offset, &vmin, &vdel, &nodeid);  // Check if there is a brick node ahead of us 

		if (brick_node != 0x0) { //We have found a brick node in ray direction.

			//Find the entrance and exit points in brick node   
			float diag_len = sqrtf(vdel.x * vdel.x);												//              b.y
			float3 b = rayBoxIntersect(wpos, dir, vmin, vmin + diag_len * gvdb->res[0]);			//          ____._____                                  
			wpos += dir * b.x;																		//		    |  /      |
			float3 brick_pos = (wpos - vmin) / vdel;												//          | / dir   |
			brick_pos += dir * 0.001;																//	    b.x |/        |
																									//          |_________|

			float3 atlas_pos = make_float3(brick_node->mValue);					// Atlas space position of brick node
			
			// ray march brick
			for (int iter = 0; iter < MAX_ITER && in_brick(gvdb, brick_pos); iter++) { 
			
				density += tex3D<float>(gvdb->volIn[chan], brick_pos.x + atlas_pos.x, brick_pos.y + atlas_pos.y, brick_pos.z + atlas_pos.z); //Sample density at voxel 
				
				brick_pos += dir * stepSize;
				wpos += dir * stepSize * vdel;
			}
			density *= vdel.x;
			

		}
		wpos += dir * stepSize;
		transmittance *= make_float3(exp(-density * stepSize));

		//TODO: get shadow transmittance and evaluate clr by albedo and extinction coefficients  
		/*  
		// calculate accumulated shadow transmittance
		float stepSizeShadow = 0.1;
		float3 shadow = getShadowTransmittance(wpos, 1.0, stepSizeShadow, extinction);

		float3 S = L * shadow * density * scattering;
		float3 sampleExtinction = make_float3(fmaxf(0.0000000001, (density * extinction).x), fmaxf(0.0000000001, (density * extinction).y), fmaxf(0.0000000001, (density * extinction).z));
		float3 Sint = (S - S * exp3(-sampleExtinction * stepSize)) / sampleExtinction;
		scatteredLuminance += transmittance * Sint;

		// Evaluate transmittance to view independentely
		transmittance *= exp3(-sampleExtinction * stepSize);
		*/

	}
	transmittance = make_float3(fminf(fmaxf(transmittance.x, 0.001), 1.0f));

	clr = make_float4(transmittance, (1- transmittance.x) * 0.1);

}



extern "C" __global__ void pathTrace(VDBInfo* gvdb, uchar chan, uchar4* outBuf) {


	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= scn.width || y >= scn.height) return;

	float3 rdir = normalize(getViewRay((float(x) + 0.5) / scn.width, (float(y) + 0.5) / scn.height));
	float3 hit = make_float3(NOHIT, NOHIT, NOHIT);
	float4 clr = make_float4(1.0f, 1.0f, 1.0f, 1.0f);

	RayCast(gvdb, chan, scn.campos, rdir, hit, clr);

	outBuf[y*scn.width + x] = make_uchar4(clr.x * 255, clr.y * 255, clr.z * 255 , 255);


}